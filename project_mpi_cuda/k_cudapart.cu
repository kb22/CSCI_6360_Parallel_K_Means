#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#define Y 3					
#define K 4				//NUMBERR OF CLUSTERS


// This kermel is used to find the index of the centroid that is nearest to each pixel.
//Ech thread is responsible for a particular pixel

__global__
void findclosestcentroids(double* num, double* centroids_c, int* idx, int each_chunk){

	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	for(int i=index; i<each_chunk; i+=stride){
		
		int x=index+offset*stride;
		int j, l, min_ind; 
		double sum, dist[K],min_dist;
		
		for (j=0;j<K;j++){
			
			sum=0;
			for (l=0;l<Y;l++){

					sum=sum+(*(num+x*Y+l)-*(centroids_c+j*Y+l))*(*(num+x*Y+l)-*(centroids_c+j*Y+l));

			}
			dist[j]=sqrt(sum);
		}
		min_dist=dist[0];
		min_ind=0;
		for (j=0; j<K; j++){
			
			if (dist[j]<min_dist) {

				min_dist=dist[j];
				min_ind=j;

			}
		}
		*(idx+x)=min_ind;
		offset++;
	}
	
}


// This kernel is launched  to update the centroids in each iteration. this is basically a reduction function where the
// mean of all the data points belonging to one cluster is calculated.

__global__
void computeCentroids(double* num, int* idx, double* centroids_c, int each_chunk){

	
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	
	int  m, j, l, count;
	double sum[Y]; 					//for(i=0;i<Y;i++) sum[i]=0.0;//is it reqd ?
	for(int i=index; i<K; i+=stride){

		int x=index+offset*stride;
		count=0;
		for(m=0;m<Y;m++) sum[m]=0.0;

		for(j =0; j<each_chunk; j++){

			if(idx[j]==x){

					count++;
					for (l=0;l<Y;l++){

						sum[l]=sum[l]+ *(num+j*Y+l);
					
					}
			
			}

		}
		if (count==0) continue;
		//printf("Counts is %d \n", count);
		for (l=0;l<Y;l++){

			*(centroids_c+x*Y+l)=sum[l]/count;					
		}
	}

}


//Kernel that performs the repalcement of each pixel in the image by the centroid it is closest to. 
//This is basically the step that quantizes the image.	


__global__
void assign_thru_gpu(double* num, double* centroids_c, int* idx, int each_chunk){
	
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread number enters the loop the next time, as the thread id will be same
	for(int i=index; i<each_chunk; i+=stride){
		
		int x=index+offset*stride;
		int i, j, k ; 

		for (k=0;k<K;k++){

			if (idx[x]==k){

					for (j=0;j<Y;j++){			
						*(num+x*Y+j)=*(centroids_c+k*Y+j);
					}
			}
				
		}
		offset++;
	}

}



//Assignment of each CUDA device to a particular rank

extern "C" void cuda_init(int each_chunk, int myrank, int numranks){

	int cudaDeviceCount=-1;
	hipError_t cE;
	//Check if cuda device exists and get the number of  working cuda devices
    if ((cE=hipGetDeviceCount( &cudaDeviceCount))!=hipSuccess){
	    printf("Unable to determine cuda Device count, error is %d count is %d \n", cE, cudaDeviceCount);
	    exit (-1);
	}
	
	//Set cuda device for each MPI rank 	
	if ((cE=hipSetDevice(myrank%cudaDeviceCount))!=hipSuccess){
	    printf("Unable to have rank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
	    exit (-1);
	}

	extern double* num;
	extern double* centroids_c;
	extern double* centroids_cresult;
	extern int* idx;

	num=NULL;
	centroids_c=NULL;
	centroids_cresult=NULL;
	idx=NULL; 

	hipMallocManaged( &num, sizeof(double)*each_chunk*Y);
    hipMallocManaged( &centroids_c, sizeof(double)*K*Y);
    hipMallocManaged( &centroids_cresult, sizeof(double)*K*Y);
	hipMallocManaged(&idx, sizeof(int)*each_chunk);


}
    
//function that initiates kernel launches from the main function.

extern "C" void k_means_kernel_launch(double* num, double* centroids_c, int* idx, int each_chunk, int n_blocks, int no_of_threads){
	
	int cudaDeviceCount;
	hipError_t cE1,cE2, cE3;

	findclosestcentroids<<< n_blocks, no_of_threads>>>(num, centroids_c, idx, each_chunk);
		
	cE1=hipGetDeviceCount( &cudaDeviceCount);
	cE2=hipDeviceSynchronize();
	//printf("The two errors are %d %d \n",cE1,cE2);
	//const char* x_err=hipGetErrorString (cE2);
	//printf("%s \n",x_err); 

	computeCentroids<<<1, 32>>>(num, &idx[0], centroids_c,each_chunk);

	cE3=hipDeviceSynchronize();
	//printf("The error is %d\n",cE3);
	//x_err=hipGetErrorString (cE3);
	//printf("%s \n",x_err); 

}

//Repalcement of each pixel in the image by the centroid it is closest to. This is the step that quantizes the image.	

extern "C" void assign(double* num, double* centroids_c, int* idx, int each_chunk, int n_blocks, int no_of_threads){
	
	assign_thru_gpu<<<n_blocks, no_of_threads>>>(num, centroids_c, idx, each_chunk);
	hipDeviceSynchronize();

}


//Freeing the dynamic memory 

extern "C" void cuda_free(double* num, double* centroids_c, double* centroids_cresult, int* idx){
	
	hipFree(num);
	hipFree(centroids_c);
	hipFree(centroids_cresult);
	hipFree(idx);

}
